#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/ATen.h>

#include "pointop.h"
#include "checks.h"


template <typename scalar_t>
__device__ __forceinline__ scalar_t activation(scalar_t x) {
    return x < 0.0f ? 0.0f : x; 
}

// only works for channel size:
// first: is it a competetive implementation? benchmark this first;
// if yes, implemt masked version; does that provide benefit?
// if yes, yaay. if no, cry.


// assuems C,H,W format
template <typename scalar_t, int C_PER_BLOCK=3, int H_PER_BLOCK=3, int W_PER_BLOCK=3, int WARP_SIZE=32>
__global__ void activation_increment_kernel(
    scalar_t *__restrict__  X,
    scalar_t const *__restrict__ in_incr,
    scalar_t * __restrict__ out_incr,  // expect a zero tenor, out
    dim const X_dim
){
    return;
    // int const warp_idx = threadIdx.x/WARP_SIZE;
    int const lane_idx = threadIdx.x%WARP_SIZE;
    int const block_idx = blockIdx.x;

    int const H_up = divup(X_dim.H, H_PER_BLOCK);
    int const W_up = divup(X_dim.W, W_PER_BLOCK);
    int const HW_up = H_up*W_up;

    int const c_in_start = block_idx/HW_up;
    int const c_in_end = min(X_dim.C, c_in_start + C_PER_BLOCK);

    int const w = W_PER_BLOCK*(block_idx%W_up) + lane_idx%W_PER_BLOCK;
    int const h = H_PER_BLOCK*(block_idx/W_up) + lane_idx/W_PER_BLOCK;

    // out of bounds
    if(lane_idx >= H_PER_BLOCK*W_PER_BLOCK || h > X_dim.H || w > X_dim.W)
        return;

    int const px_offs = h*X_dim.W + w;

    // for(int i = 0; i < )
    for(int c = c_in_start; c < c_in_end ; c += 1){
        int x_id = c*X_dim.H*X_dim.W + px_offs;
        scalar_t* reserve = &X[x_id];
        scalar_t const * incr = &in_incr[x_id];
        scalar_t const full = *reserve + *incr;
        out_incr[x_id] = activation(full) - activation(*reserve);
        *reserve = full;
    }
}

template <typename scalar_t, int C_PER_BLOCK=3, int H_PER_BLOCK=3, int W_PER_BLOCK=3>
void activation_increment_cuda(
    torch::Tensor &X,
    torch::Tensor const &in_incr,
    torch::Tensor &out_incr  // expect a zero tensor
){
    auto X_dim = dim(X.sizes());

    // per block function: 3*3*C_PER_BLOCK
    int const H_up = divup(X_dim.H, H_PER_BLOCK);
    int const W_up = divup(X_dim.W, W_PER_BLOCK);
    int const C_up = divup(X_dim.C, C_PER_BLOCK);

    int const blocks = H_up*C_up*W_up;
    int const threads = 32;

    activation_increment_kernel<scalar_t, C_PER_BLOCK, H_PER_BLOCK, W_PER_BLOCK><<<blocks, threads>>>(
        X.data_ptr<scalar_t>(), 
        in_incr.data_ptr<scalar_t>(), 
        out_incr.data_ptr<scalar_t>(), 
        X_dim
    );

    CUDA_CHECK_ERRORS();
}


void activation_increment_cuda_wrapper(
    torch::Tensor &X,
    torch::Tensor const &in_incr,
    torch::Tensor &out_incr  // expect a zero tensor
){
    activation_increment_cuda<float>(
        X,
        in_incr,
        out_incr  // expect a zero tensor
    );
}

